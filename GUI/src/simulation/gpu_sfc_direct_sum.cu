#include "hip/hip_runtime.h"
#include "../../include/simulation/gpu_sfc_direct_sum.cuh"
#include "../../include/sfc/morton.cuh"
#include <iostream>

/**
 * @brief CUDA kernel for direct force calculation between all body pairs, with SFC ordering support
 *
 * This kernel computes the gravitational forces between all pairs of bodies
 * using the Direct Sum approach (O(n²) complexity), with support for SFC-ordered indices.
 *
 * @param bodies Array of body structures
 * @param orderedIndices Array of SFC-ordered indices (can be nullptr if SFC not used)
 * @param useSFC Flag indicating whether to use SFC ordering
 * @param nBodies Number of bodies in the simulation
 */
__global__ void SFCDirectSumForceKernel(Body *bodies, int *orderedIndices, bool useSFC, int nBodies)
{
    // Reduced size shared memory arrays
    __shared__ Vector sharedPos[256];  // Reduced from BLOCK_SIZE
    __shared__ double sharedMass[256]; // Reduced from BLOCK_SIZE

    // Get global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    // Get the real body index when using SFC ordering
    int realBodyIndex = (useSFC && orderedIndices != nullptr) ? orderedIndices[i] : i;

    // Load data only if the index is valid
    Vector myPos = Vector(0, 0, 0);
    Vector myVel = Vector(0, 0, 0);
    Vector myAcc = Vector(0, 0, 0);
    double myMass = 0.0;
    bool isDynamic = false;

    if (i < nBodies)
    {
        myPos = bodies[realBodyIndex].position;
        myVel = bodies[realBodyIndex].velocity;
        myMass = bodies[realBodyIndex].mass;
        isDynamic = bodies[realBodyIndex].isDynamic;
    }

    // Reduce computation with tiling approach
    const int tileSize = 256; // Smaller tile size for better occupancy

    // Process all tiles
    for (int tile = 0; tile < (nBodies + tileSize - 1) / tileSize; ++tile)
    {
        // Load this tile to shared memory
        int idx = tile * tileSize + tx;

        // Only load valid data to shared memory
        if (tx < tileSize)
        { // Ensure we don't exceed array size
            if (idx < nBodies)
            {
                // When using SFC ordering, get the real body index
                int tileBodyIndex = (useSFC && orderedIndices != nullptr) ? orderedIndices[idx] : idx;
                sharedPos[tx] = bodies[tileBodyIndex].position;
                sharedMass[tx] = bodies[tileBodyIndex].mass;
            }
            else
            {
                sharedPos[tx] = Vector(0, 0, 0);
                sharedMass[tx] = 0.0;
            }
        }

        __syncthreads();

        // Calculate force only for valid and dynamic bodies
        if (i < nBodies && isDynamic)
        {
            // Limit the loop to the real tile size
            int tileLimit = min(tileSize, nBodies - tile * tileSize);

            for (int j = 0; j < tileLimit; ++j)
            {
                int jBody = tile * tileSize + j;

                // Avoid self-interaction
                if (jBody != i)
                {
                    // Distance vector
                    double rx = sharedPos[j].x - myPos.x;
                    double ry = sharedPos[j].y - myPos.y;
                    double rz = sharedPos[j].z - myPos.z;

                    // Distance squared with softening
                    double distSqr = rx * rx + ry * ry + rz * rz + E * E;
                    double dist = sqrt(distSqr);

                    // Apply force only if above collision threshold
                    if (dist >= COLLISION_TH)
                    {
                        double forceMag = (GRAVITY * myMass * sharedMass[j]) / (dist * distSqr);

                        // Accumulate acceleration
                        myAcc.x += rx * forceMag / myMass;
                        myAcc.y += ry * forceMag / myMass;
                        myAcc.z += rz * forceMag / myMass;
                    }
                }
            }
        }

        __syncthreads();
    }

    // Update the body only if valid and dynamic
    if (i < nBodies && isDynamic)
    {
        // Save acceleration
        bodies[realBodyIndex].acceleration = myAcc;

        // Update velocity
        myVel.x += myAcc.x * DT;
        myVel.y += myAcc.y * DT;
        myVel.z += myAcc.z * DT;
        bodies[realBodyIndex].velocity = myVel;

        // Update position
        myPos.x += myVel.x * DT;
        myPos.y += myVel.y * DT;
        myPos.z += myVel.z * DT;
        bodies[realBodyIndex].position = myPos;
    }
}

SFCGPUDirectSum::SFCGPUDirectSum(int numBodies, bool useSpaceFillingCurve,
                                 int initialReorderFreq, BodyDistribution dist, unsigned int seed)
    : GPUDirectSum(numBodies, dist, seed),
      useSFC(useSpaceFillingCurve),
      sorter(nullptr),
      d_orderedIndices(nullptr),
      reorderFrequency(initialReorderFreq),
      iterationCounter(0)
{
    if (useSFC)
    {
        sorter = new sfc::BodySorter(numBodies);
    }

    // Initialize domain bounds to invalid values to force update
    minBound = Vector(INFINITY, INFINITY, INFINITY);
    maxBound = Vector(-INFINITY, -INFINITY, -INFINITY);

    std::cout << "SFC GPU Direct Sum Simulation created with " << numBodies << " bodies." << std::endl;
    if (useSFC)
    {
        std::cout << "Space-Filling Curve ordering enabled with reorder frequency "
                  << reorderFrequency << std::endl;
    }
}

SFCGPUDirectSum::~SFCGPUDirectSum()
{
    if (sorter)
    {
        delete sorter;
        sorter = nullptr;
    }

    // The d_orderedIndices is managed by the sorter, so we don't free it here
}

void SFCGPUDirectSum::updateBoundingBox()
{
    // We need to compute bounding box for SFC ordering
    // Using a simple kernel launch or by copying data to host

    // Temporary solution: allocate host memory and copy bodies
    Body *tempBodies = new Body[nBodies];
    CHECK_CUDA_ERROR(hipMemcpy(tempBodies, d_bodies, nBodies * sizeof(Body), hipMemcpyDeviceToHost));

    // Find min and max bounds
    minBound = Vector(INFINITY, INFINITY, INFINITY);
    maxBound = Vector(-INFINITY, -INFINITY, -INFINITY);

    for (int i = 0; i < nBodies; i++)
    {
        Vector pos = tempBodies[i].position;

        // Update minimum bounds
        minBound.x = std::min(minBound.x, pos.x);
        minBound.y = std::min(minBound.y, pos.y);
        minBound.z = std::min(minBound.z, pos.z);

        // Update maximum bounds
        maxBound.x = std::max(maxBound.x, pos.x);
        maxBound.y = std::max(maxBound.y, pos.y);
        maxBound.z = std::max(maxBound.z, pos.z);
    }

    // Add padding to avoid edge issues
    double padding = std::max(1.0e10, (maxBound.x - minBound.x) * 0.01);
    minBound.x -= padding;
    minBound.y -= padding;
    minBound.z -= padding;
    maxBound.x += padding;
    maxBound.y += padding;
    maxBound.z += padding;

    // Cleanup
    delete[] tempBodies;
}

void SFCGPUDirectSum::orderBodiesBySFC()
{
    if (!useSFC || !sorter)
    {
        d_orderedIndices = nullptr;
        return;
    }

    // Update bounds for SFC calculation
    updateBoundingBox();

    // Get indices ordered by SFC
    d_orderedIndices = sorter->sortBodies(d_bodies, minBound, maxBound);
}

void SFCGPUDirectSum::computeForces()
{
    // Measure execution time
    CudaTimer timer(metrics.forceTimeMs);

    // Launch kernel with SFC support
    int blockSize = 256; // Reduced block size for better occupancy
    int gridSize = (nBodies + blockSize - 1) / blockSize;

    SFCDirectSumForceKernel<<<gridSize, blockSize>>>(d_bodies, d_orderedIndices, useSFC, nBodies);
    CHECK_LAST_CUDA_ERROR();
}

void SFCGPUDirectSum::update()
{
    // Ensure initialization
    checkInitialization();

    // Measure total execution time
    CudaTimer timer(metrics.totalTimeMs);

    // Reset unused metrics
    metrics.resetTimeMs = 0.0f;  // Not used in Direct Sum
    metrics.bboxTimeMs = 0.0f;   // Used internally in SFC ordering
    metrics.octreeTimeMs = 0.0f; // Not used in Direct Sum

    // Apply SFC ordering if enabled
    if (useSFC)
    {
        // Increment the iteration counter
        iterationCounter++;

        // Only reorder if it's time based on the frequency or this is the first iteration
        if (iterationCounter >= reorderFrequency || iterationCounter == 1)
        {
            // Reset the counter
            iterationCounter = 0;

            // Perform the SFC ordering
            orderBodiesBySFC();
        }
    }
    else
    {
        d_orderedIndices = nullptr;
    }

    // Compute forces and update positions
    computeForces();
}