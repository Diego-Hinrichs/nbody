#include "hip/hip_runtime.h"
#include "../../include/simulation/implementations/gpu/sfc_variants.cuh"
#include "../../include/sfc/sfc_framework.cuh"
#include <iostream>
#include <algorithm>

SFCBarnesHut::SFCBarnesHut(int numBodies, bool useSpaceFillingCurve,
                           SFCOrderingMode initialOrderingMode, int initialReorderFreq,
                           BodyDistribution dist, unsigned int seed)
    : BarnesHut(numBodies, dist, seed),
      useSFC(useSpaceFillingCurve),
      bodySorter(nullptr),
      octantSorter(nullptr),
      d_orderedIndices(nullptr),
      d_octantIndices(nullptr),
      orderingMode(initialOrderingMode),
      reorderFrequency(initialReorderFreq),
      iterationCounter(0),
      curveType(sfc::CurveType::MORTON) // Default to Morton curve
{
    if (useSFC)
    {
        // Create sorters with appropriate curve type
        bodySorter = new sfc::BodySorter(numBodies, curveType);
        octantSorter = new sfc::OctantSorter(MAX_NODES, curveType);
    }

    // Initialize domain bounds to invalid values to force update
    minBound = Vector(INFINITY, INFINITY, INFINITY);
    maxBound = Vector(-INFINITY, -INFINITY, -INFINITY);

    std::cout << "SFC Barnes-Hut Simulation created with " << numBodies << " bodies." << std::endl;
    if (useSFC)
    {
        std::cout << "Space-Filling Curve ordering enabled with "
                  << (orderingMode == SFCOrderingMode::PARTICLES ? "particle" : "octant")
                  << " ordering and reorder frequency " << reorderFrequency << std::endl;
    }
}

SFCBarnesHut::~SFCBarnesHut()
{
    if (bodySorter)
    {
        delete bodySorter;
        bodySorter = nullptr;
    }

    if (octantSorter)
    {
        delete octantSorter;
        octantSorter = nullptr;
    }

    // Note: d_orderedIndices and d_octantIndices are managed by the sorters
}

void SFCBarnesHut::setCurveType(sfc::CurveType type)
{
    if (type != curveType)
    {
        curveType = type;

        // Update sorters with new curve type
        if (bodySorter)
            bodySorter->setCurveType(type);

        if (octantSorter)
            octantSorter->setCurveType(type);

        // Force reordering on next update
        iterationCounter = reorderFrequency;
    }
}

void SFCBarnesHut::updateBoundingBox()
{
    // Copy root node to get current bounding box
    Node rootNode;
    CHECK_CUDA_ERROR(hipMemcpy(&rootNode, d_nodes, sizeof(Node), hipMemcpyDeviceToHost));

    // Update domain bounds
    minBound = rootNode.topLeftFront;
    maxBound = rootNode.botRightBack;

    // Add a small padding to avoid edge cases
    double padding = std::max(1.0e10, (maxBound.x - minBound.x) * 0.01);
    minBound.x -= padding;
    minBound.y -= padding;
    minBound.z -= padding;
    maxBound.x += padding;
    maxBound.y += padding;
    maxBound.z += padding;
}

void SFCBarnesHut::orderBodiesBySFC()
{
    if (!useSFC || !bodySorter)
    {
        d_orderedIndices = nullptr;
        return;
    }

    // Update bounds for SFC calculation
    updateBoundingBox();

    // Get indices ordered by SFC
    d_orderedIndices = bodySorter->sortBodies(d_bodies, minBound, maxBound);
}

void SFCBarnesHut::orderOctantsBySFC(Node *nodes, int nNodes)
{
    if (!useSFC || !octantSorter)
    {
        d_octantIndices = nullptr;
        return;
    }

    // Update domain bounds
    updateBoundingBox();

    // Get indices ordered by SFC
    d_octantIndices = octantSorter->sortOctants(nodes, minBound, maxBound);
}

// Override constructOctree from the base class
void SFCBarnesHut::constructOctree()
{
    // Measure execution time
    CudaTimer timer(metrics.octreeTimeMs);

    // Determine if we're using octant ordering mode
    bool useOctantOrder = (useSFC && orderingMode == SFCOrderingMode::OCTANTS && d_octantIndices != nullptr);

    // Launch octree construction kernel with appropriate ordering
    BuildOptimizedOctTree(d_nodes, d_bodies, d_tempBodies,
                          d_orderedIndices, useSFC && orderingMode == SFCOrderingMode::PARTICLES,
                          d_octantIndices, useOctantOrder,
                          nNodes, nBodies, leafLimit);
    CHECK_LAST_CUDA_ERROR();
}

void SFCBarnesHut::update()
{
    // Ensure initialization
    checkInitialization();

    // Measure total execution time
    CudaTimer timer(metrics.totalTimeMs);

    // Execute the Barnes-Hut algorithm steps with SFC enhancement
    resetOctree();
    computeBoundingBox();

    // Apply SFC ordering based on mode and reordering frequency
    if (useSFC)
    {
        // Increment the iteration counter
        iterationCounter++;

        // Only reorder if it's time based on the frequency or this is the first iteration
        if (iterationCounter >= reorderFrequency || iterationCounter == 1)
        {
            // Reset the counter
            iterationCounter = 0;

            // Apply appropriate ordering based on the selected mode
            if (orderingMode == SFCOrderingMode::PARTICLES)
            {
                orderBodiesBySFC();
                d_octantIndices = nullptr; // Not using octant ordering
            }
            else // OCTANTS mode
            {
                // For octant ordering, we need to:
                // 1. Build tree first (with no ordering)
                d_orderedIndices = nullptr; // Not using particle ordering

                // Use base class's constructOctree method to build initial tree
                BarnesHut::constructOctree();

                // 2. Compute SFC codes for octants
                orderOctantsBySFC(d_nodes, nNodes);

                // 3. Now build the octree again, but with octant ordering
                constructOctree();
            }
        }
    }
    else
    {
        d_orderedIndices = nullptr;
        d_octantIndices = nullptr;
    }

    // Construct tree if not already built
    if (orderingMode != SFCOrderingMode::OCTANTS || !useSFC || iterationCounter != 0)
    {
        constructOctree();
    }

    // Compute forces
    computeForces();
}

/**
 * @brief CUDA kernel for direct force calculation between all body pairs, with SFC ordering support
 *
 * This kernel computes the gravitational forces between all pairs of bodies
 * using the Direct Sum approach (O(n²) complexity), with support for SFC-ordered indices.
 *
 * @param bodies Array of body structures
 * @param orderedIndices Array of SFC-ordered indices (can be nullptr if SFC not used)
 * @param useSFC Flag indicating whether to use SFC ordering
 * @param nBodies Number of bodies in the simulation
 */
__global__ void SFCDirectSumForceKernel(Body *bodies, int *orderedIndices, bool useSFC, int nBodies)
{
    // Reduced size shared memory arrays
    __shared__ Vector sharedPos[256];  // Reduced from BLOCK_SIZE
    __shared__ double sharedMass[256]; // Reduced from BLOCK_SIZE

    // Get global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    // Get the real body index when using SFC ordering
    int realBodyIndex = (useSFC && orderedIndices != nullptr) ? orderedIndices[i] : i;

    // Load data only if the index is valid
    Vector myPos = Vector(0, 0, 0);
    Vector myVel = Vector(0, 0, 0);
    Vector myAcc = Vector(0, 0, 0);
    double myMass = 0.0;
    bool isDynamic = false;

    if (i < nBodies)
    {
        myPos = bodies[realBodyIndex].position;
        myVel = bodies[realBodyIndex].velocity;
        myMass = bodies[realBodyIndex].mass;
        isDynamic = bodies[realBodyIndex].isDynamic;
    }

    // Reduce computation with tiling approach
    const int tileSize = 256; // Smaller tile size for better occupancy

    // Process all tiles
    for (int tile = 0; tile < (nBodies + tileSize - 1) / tileSize; ++tile)
    {
        // Load this tile to shared memory
        int idx = tile * tileSize + tx;

        // Only load valid data to shared memory
        if (tx < tileSize)
        { // Ensure we don't exceed array size
            if (idx < nBodies)
            {
                // When using SFC ordering, get the real body index
                int tileBodyIndex = (useSFC && orderedIndices != nullptr) ? orderedIndices[idx] : idx;
                sharedPos[tx] = bodies[tileBodyIndex].position;
                sharedMass[tx] = bodies[tileBodyIndex].mass;
            }
            else
            {
                sharedPos[tx] = Vector(0, 0, 0);
                sharedMass[tx] = 0.0;
            }
        }

        __syncthreads();

        // Calculate force only for valid and dynamic bodies
        if (i < nBodies && isDynamic)
        {
            // Limit the loop to the real tile size
            int tileLimit = min(tileSize, nBodies - tile * tileSize);

            for (int j = 0; j < tileLimit; ++j)
            {
                int jBody = tile * tileSize + j;

                // Avoid self-interaction
                if (jBody != i)
                {
                    // Distance vector
                    double rx = sharedPos[j].x - myPos.x;
                    double ry = sharedPos[j].y - myPos.y;
                    double rz = sharedPos[j].z - myPos.z;

                    // Distance squared with softening
                    double distSqr = rx * rx + ry * ry + rz * rz + E * E;
                    double dist = sqrt(distSqr);

                    // Apply force only if above collision threshold
                    if (dist >= COLLISION_TH)
                    {
                        double forceMag = (GRAVITY * myMass * sharedMass[j]) / (dist * distSqr);

                        // Accumulate acceleration
                        myAcc.x += rx * forceMag / myMass;
                        myAcc.y += ry * forceMag / myMass;
                        myAcc.z += rz * forceMag / myMass;
                    }
                }
            }
        }

        __syncthreads();
    }

    // Update the body only if valid and dynamic
    if (i < nBodies && isDynamic)
    {
        // Save acceleration
        bodies[realBodyIndex].acceleration = myAcc;

        // Update velocity
        myVel.x += myAcc.x * DT;
        myVel.y += myAcc.y * DT;
        myVel.z += myAcc.z * DT;
        bodies[realBodyIndex].velocity = myVel;

        // Update position
        myPos.x += myVel.x * DT;
        myPos.y += myVel.y * DT;
        myPos.z += myVel.z * DT;
        bodies[realBodyIndex].position = myPos;
    }
}

SFCGPUDirectSum::SFCGPUDirectSum(int numBodies, bool useSpaceFillingCurve,
                                 int initialReorderFreq, BodyDistribution dist, unsigned int seed)
    : GPUDirectSum(numBodies, dist, seed),
      useSFC(useSpaceFillingCurve),
      sorter(nullptr),
      d_orderedIndices(nullptr),
      curveType(sfc::CurveType::MORTON), // Default to Morton curve
      reorderFrequency(initialReorderFreq),
      iterationCounter(0)
{
    if (useSFC)
    {
        sorter = new sfc::BodySorter(numBodies, curveType);
    }

    // Initialize domain bounds to invalid values to force update
    minBound = Vector(INFINITY, INFINITY, INFINITY);
    maxBound = Vector(-INFINITY, -INFINITY, -INFINITY);

    std::cout << "SFC GPU Direct Sum Simulation created with " << numBodies << " bodies." << std::endl;
    if (useSFC)
    {
        std::cout << "Space-Filling Curve ordering enabled with reorder frequency "
                  << reorderFrequency << std::endl;
    }
}

SFCGPUDirectSum::~SFCGPUDirectSum()
{
    if (sorter)
    {
        delete sorter;
        sorter = nullptr;
    }

    // The d_orderedIndices is managed by the sorter, so we don't free it here
}

void SFCGPUDirectSum::setCurveType(sfc::CurveType type)
{
    if (type != curveType)
    {
        curveType = type;

        // Update sorter with new curve type
        if (sorter)
            sorter->setCurveType(type);

        // Force reordering on next update
        iterationCounter = reorderFrequency;
    }
}

void SFCGPUDirectSum::updateBoundingBox()
{
    // We need to compute bounding box for SFC ordering
    // Using a simple kernel launch or by copying data to host

    // Temporary solution: allocate host memory and copy bodies
    Body *tempBodies = new Body[nBodies];
    CHECK_CUDA_ERROR(hipMemcpy(tempBodies, d_bodies, nBodies * sizeof(Body), hipMemcpyDeviceToHost));

    // Find min and max bounds
    minBound = Vector(INFINITY, INFINITY, INFINITY);
    maxBound = Vector(-INFINITY, -INFINITY, -INFINITY);

    for (int i = 0; i < nBodies; i++)
    {
        Vector pos = tempBodies[i].position;

        // Update minimum bounds
        minBound.x = std::min(minBound.x, pos.x);
        minBound.y = std::min(minBound.y, pos.y);
        minBound.z = std::min(minBound.z, pos.z);

        // Update maximum bounds
        maxBound.x = std::max(maxBound.x, pos.x);
        maxBound.y = std::max(maxBound.y, pos.y);
        maxBound.z = std::max(maxBound.z, pos.z);
    }

    // Add padding to avoid edge issues
    double padding = std::max(1.0e10, (maxBound.x - minBound.x) * 0.01);
    minBound.x -= padding;
    minBound.y -= padding;
    minBound.z -= padding;
    maxBound.x += padding;
    maxBound.y += padding;
    maxBound.z += padding;

    // Cleanup
    delete[] tempBodies;
}

void SFCGPUDirectSum::orderBodiesBySFC()
{
    if (!useSFC || !sorter)
    {
        d_orderedIndices = nullptr;
        return;
    }

    // Update bounds for SFC calculation
    updateBoundingBox();

    // Get indices ordered by SFC
    d_orderedIndices = sorter->sortBodies(d_bodies, minBound, maxBound);
}

void SFCGPUDirectSum::computeForces()
{
    // Measure execution time
    CudaTimer timer(metrics.forceTimeMs);

    // Launch kernel with SFC support
    int blockSize = 256; // Reduced block size for better occupancy
    int gridSize = (nBodies + blockSize - 1) / blockSize;

    SFCDirectSumForceKernel<<<gridSize, blockSize>>>(d_bodies, d_orderedIndices, useSFC, nBodies);
    CHECK_LAST_CUDA_ERROR();
}

void SFCGPUDirectSum::update()
{
    // Ensure initialization
    checkInitialization();

    // Measure total execution time
    CudaTimer timer(metrics.totalTimeMs);

    // Reset unused metrics
    metrics.resetTimeMs = 0.0f;  // Not used in Direct Sum
    metrics.bboxTimeMs = 0.0f;   // Used internally in SFC ordering
    metrics.octreeTimeMs = 0.0f; // Not used in Direct Sum

    // Apply SFC ordering if enabled
    if (useSFC)
    {
        // Increment the iteration counter
        iterationCounter++;

        // Only reorder if it's time based on the frequency or this is the first iteration
        if (iterationCounter >= reorderFrequency || iterationCounter == 1)
        {
            // Reset the counter
            iterationCounter = 0;

            // Perform the SFC ordering
            orderBodiesBySFC();
        }
    }
    else
    {
        d_orderedIndices = nullptr;
    }

    // Compute forces and update positions
    computeForces();
}