#include "hip/hip_runtime.h"
#include "../../include/common/types.cuh"
#include "../../include/common/constants.cuh"

/**
 * @brief Compute the bounding box for the entire simulation domain
 *
 * This kernel uses a parallel reduction approach to find the minimum and
 * maximum coordinates of all bodies, defining the simulation domain.
 *
 * @param nodes Array of octree nodes (only updates the root node)
 * @param bodies Array of bodies
 * @param mutex Mutex for synchronization
 * @param nBodies Number of bodies in the simulation
 */
__global__ void ComputeBoundingBoxKernel(Node *nodes, Body *bodies, int *mutex, int nBodies)
{
    // Shared memory for parallel reduction of each dimension
    __shared__ double topLeftFrontX[BLOCK_SIZE];
    __shared__ double topLeftFrontY[BLOCK_SIZE];
    __shared__ double topLeftFrontZ[BLOCK_SIZE];
    __shared__ double botRightBackX[BLOCK_SIZE];
    __shared__ double botRightBackY[BLOCK_SIZE];
    __shared__ double botRightBackZ[BLOCK_SIZE];

    int tx = threadIdx.x;
    int b = blockIdx.x * blockDim.x + tx;

    // Initialize with extreme values
    topLeftFrontX[tx] = INFINITY;  // Min X
    topLeftFrontY[tx] = INFINITY;  // Min Y
    topLeftFrontZ[tx] = INFINITY;  // Min Z
    botRightBackX[tx] = -INFINITY; // Max X
    botRightBackY[tx] = -INFINITY; // Max Y
    botRightBackZ[tx] = -INFINITY; // Max Z

    __syncthreads();

    // Load body data if within range
    if (b < nBodies)
    {
        Body body = bodies[b];
        topLeftFrontX[tx] = body.position.x;
        topLeftFrontY[tx] = body.position.y;
        topLeftFrontZ[tx] = body.position.z;

        botRightBackX[tx] = body.position.x;
        botRightBackY[tx] = body.position.y;
        botRightBackZ[tx] = body.position.z;
    }

    // Parallel reduction to find min/max values
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        __syncthreads();
        if (tx < s)
        {
            // Min reduction for top-left-front
            topLeftFrontX[tx] = fmin(topLeftFrontX[tx], topLeftFrontX[tx + s]);
            topLeftFrontY[tx] = fmin(topLeftFrontY[tx], topLeftFrontY[tx + s]);
            topLeftFrontZ[tx] = fmin(topLeftFrontZ[tx], topLeftFrontZ[tx + s]);

            // Max reduction for bottom-right-back
            botRightBackX[tx] = fmax(botRightBackX[tx], botRightBackX[tx + s]);
            botRightBackY[tx] = fmax(botRightBackY[tx], botRightBackY[tx + s]);
            botRightBackZ[tx] = fmax(botRightBackZ[tx], botRightBackZ[tx + s]);
        }
    }

    // Update root node with mutex to avoid race conditions
    if (tx == 0)
    {
        // Wait until mutex is available
        while (atomicCAS(mutex, 0, 1) != 0)
        {
        }

        // Update bounds with a margin for numerical stability
        // Update minimum bounds (top-left-front corner)
        nodes[0].topLeftFront.x = fmin(nodes[0].topLeftFront.x, topLeftFrontX[0] - 1.0e10);
        nodes[0].topLeftFront.y = fmin(nodes[0].topLeftFront.y, topLeftFrontY[0] - 1.0e10);
        nodes[0].topLeftFront.z = fmin(nodes[0].topLeftFront.z, topLeftFrontZ[0] - 1.0e10);

        // Update maximum bounds (bottom-right-back corner)
        nodes[0].botRightBack.x = fmax(nodes[0].botRightBack.x, botRightBackX[0] + 1.0e10);
        nodes[0].botRightBack.y = fmax(nodes[0].botRightBack.y, botRightBackY[0] + 1.0e10);
        nodes[0].botRightBack.z = fmax(nodes[0].botRightBack.z, botRightBackZ[0] + 1.0e10);

        // Release mutex
        atomicExch(mutex, 0);
    }
}