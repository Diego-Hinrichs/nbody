#include "hip/hip_runtime.h"
#include "../../include/simulation/implementations/gpu/barnes_hut.cuh"

BarnesHut::BarnesHut(int numBodies, BodyDistribution dist, unsigned int seed)
    : SimulationBase(numBodies, dist, seed)
{
    nNodes = MAX_NODES;
    leafLimit = MAX_NODES - N_LEAF;

    // Allocate host memory for nodes
    h_nodes = new Node[nNodes];

    // Allocate device memory
    CHECK_CUDA_ERROR(hipMalloc(&d_nodes, nNodes * sizeof(Node)));
    CHECK_CUDA_ERROR(hipMalloc(&d_mutex, nNodes * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_bodiesBuffer, nBodies * sizeof(Body)));
}

BarnesHut::~BarnesHut()
{
    // Free host memory
    if (h_nodes)
    {
        delete[] h_nodes;
        h_nodes = nullptr;
    }

    // Free device memory
    if (d_nodes)
    {
        CHECK_CUDA_ERROR(hipFree(d_nodes));
        d_nodes = nullptr;
    }

    if (d_mutex)
    {
        CHECK_CUDA_ERROR(hipFree(d_mutex));
        d_mutex = nullptr;
    }

    if (d_tempBodies)
    {
        hipFree(d_tempBodies);
        d_tempBodies = nullptr;
    }

    if (d_bodiesBuffer)
    {
        CHECK_CUDA_ERROR(hipFree(d_bodiesBuffer));
        d_bodiesBuffer = nullptr;
    }
}

void BarnesHut::resetOctree()
{
    CudaTimer timer(metrics.resetTimeMs);
    int blockSize = BLOCK_SIZE;
    dim3 gridSize = ceil((float)nNodes / blockSize);
    ResetKernel<<<gridSize, blockSize>>>(d_nodes, d_mutex, nNodes, nBodies);
    CHECK_LAST_CUDA_ERROR();
}

void BarnesHut::computeBoundingBox()
{
    CudaTimer timer(metrics.bboxTimeMs);
    int blockSize = BLOCK_SIZE;
    dim3 gridSize = ceil((float)nBodies / blockSize);
    ComputeBoundingBoxKernel<<<gridSize, blockSize>>>(d_nodes, d_bodies, getOrderedIndices(), isUsingSFC(), d_mutex, nBodies);
    CHECK_LAST_CUDA_ERROR();
}

void BarnesHut::constructOctree()
{
    CudaTimer timer(metrics.octreeTimeMs);
    int blockSize = BLOCK_SIZE;
    ConstructOctTreeKernel<<<1, blockSize>>>(d_nodes, d_bodies, d_bodiesBuffer, 0, nNodes, nBodies, leafLimit);
    CHECK_LAST_CUDA_ERROR();
}

void BarnesHut::computeForces()
{
    CudaTimer timer(metrics.forceTimeMs);
    int blockSize = 32; // x cache
    dim3 gridSize = ceil((float)nBodies / blockSize);
    ComputeForceKernel<<<gridSize, blockSize>>>(d_nodes, d_bodies, nNodes, nBodies, leafLimit);
    CHECK_LAST_CUDA_ERROR();
}

void BarnesHut::update()
{
    checkInitialization();

    CudaTimer timer(metrics.totalTimeMs);
    resetOctree();
    computeBoundingBox();
    constructOctree();
    computeForces();
}