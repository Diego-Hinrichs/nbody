#include "hip/hip_runtime.h"
/*
   Copyright 2023 Hsin-Hung Wu

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include <iostream>
#include <cmath>
#include <algorithm>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include "barnesHut_kernel.cuh"
#include "constants.h"
#include "err.h"
#include "sfc_utils.cuh"
#include <vector>

// Constructor including SFC parameters
BarnesHutCuda::BarnesHutCuda(int n, SFCType sfc, OrderTarget target)
    : nBodies(n), sfcType(sfc), orderTarget(target)
{
    nNodes = MAX_NODES;
    leafLimit = MAX_NODES - N_LEAF;
    h_b = new Body[nBodies];
    h_node = new Node[nNodes];

    CHECK_CUDA_ERROR(hipMalloc(&d_b, n * sizeof(Body)));
    CHECK_CUDA_ERROR(hipMalloc(&d_node, nNodes * sizeof(Node)));
    CHECK_CUDA_ERROR(hipMalloc(&d_mutex, nNodes * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_b_buffer, n * sizeof(Body)));
}

BarnesHutCuda::~BarnesHutCuda()
{
    delete[] h_b;
    delete[] h_node;
    CHECK_CUDA_ERROR(hipFree(d_b));
    CHECK_CUDA_ERROR(hipFree(d_node));
    CHECK_CUDA_ERROR(hipFree(d_mutex));
    CHECK_CUDA_ERROR(hipFree(d_b_buffer));
}

void BarnesHutCuda::resetCUDA()
{
    int blockSize = BLOCK_SIZE;
    dim3 gridSize = ceil((float)nNodes / blockSize);
    ResetKernel<<<gridSize, blockSize>>>(d_node, d_mutex, nNodes, nBodies);
}

void BarnesHutCuda::computeBoundingBoxCUDA()
{
    int blockSize = BLOCK_SIZE;
    dim3 gridSize = ceil((float)nBodies / blockSize);
    ComputeBoundingBoxKernel<<<gridSize, blockSize>>>(d_node, d_b, d_mutex, nBodies);
}

void BarnesHutCuda::constructOctreeCUDA()
{
    int blockSize = BLOCK_SIZE;
    ConstructOctTreeKernel<<<1, blockSize>>>(d_node, d_b, d_b_buffer, 0, nNodes, nBodies, leafLimit);
}

void BarnesHutCuda::computeForceCUDA()
{
    int blockSize = 32;
    dim3 gridSize = ceil((float)nBodies / blockSize);
    ComputeForceKernel<<<gridSize, blockSize>>>(d_node, d_b, nNodes, nBodies, leafLimit);
}

// New method to apply SFC ordering
void BarnesHutCuda::applySFCOrdering()
{
    if (sfcType == NO_SFC || orderTarget == ORDER_NONE)
    {
        return; // No ordering requested
    }

    // Get the bounding box to normalize coordinates
    Vector min, max;
    // hipMemcpy(&min, &d_node[0].topLeftFront, sizeof(Vector), hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(hipMemcpy(&min, &d_node[0].topLeftFront, sizeof(Vector), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(&max, &d_node[0].botRightBack, sizeof(Vector), hipMemcpyDeviceToHost));

    if (orderTarget == ORDER_BODIES)
    {
        // Create Thrust device pointers for sorting
        thrust::device_ptr<Body> d_bodies_ptr(d_b);

        // Apply different ordering based on the selected SFC
        if (sfcType == MORTON)
        {
            // Use Morton ordering
            MortonComparator comp(min, max);
            thrust::sort(thrust::device, d_bodies_ptr, d_bodies_ptr + nBodies, comp);
        }
        else if (sfcType == HILBERT)
        {
            // Use Hilbert ordering
            HilbertComparator comp(min, max);
            thrust::sort(thrust::device, d_bodies_ptr, d_bodies_ptr + nBodies, comp);
        }
    }
    else if (orderTarget == ORDER_OCTANTS)
    {
        // Ordering octants would be more complex and would require
        // modifying the octree construction logic
        // For now, we implement a placeholder that copies the nodes to host,
        // computes SFC codes, sorts, and copies back

        // Copy nodes to host
        Node *temp_nodes = new Node[nNodes];
        CHECK_CUDA_ERROR(hipMemcpy(temp_nodes, d_node, nNodes * sizeof(Node), hipMemcpyDeviceToHost));

        // Compute SFC codes for each node
        for (int i = 0; i < nNodes; i++)
        {
            if (temp_nodes[i].start != -1 && temp_nodes[i].end != -1)
            {
                // Calculate center point of the node
                Vector center = {
                    (temp_nodes[i].topLeftFront.x + temp_nodes[i].botRightBack.x) / 2.0,
                    (temp_nodes[i].topLeftFront.y + temp_nodes[i].botRightBack.y) / 2.0,
                    (temp_nodes[i].topLeftFront.z + temp_nodes[i].botRightBack.z) / 2.0};

                // Compute SFC code based on center
                if (sfcType == MORTON)
                {
                    temp_nodes[i].sfcCode = positionToMortonCode(center, min, max);
                }
                else if (sfcType == HILBERT)
                {
                    temp_nodes[i].sfcCode = positionToHilbertCode(center, min, max);
                }
            }
        }

        // Sort nodes based on SFC code (preserving octree structure)
        // This is a simple example - real implementation would need to be more sophisticated
        // to maintain the tree structure

        // Copy back to device
        CHECK_CUDA_ERROR(hipMemcpy(d_node, temp_nodes, nNodes * sizeof(Node), hipMemcpyHostToDevice));

        delete[] temp_nodes;
    }
}

void BarnesHutCuda::setBody(int i, bool isDynamic, double mass, double radius, Vector position, Vector velocity, Vector acceleration)
{
    h_b[i].isDynamic = isDynamic;
    h_b[i].mass = mass;
    h_b[i].radius = radius;
    h_b[i].position = position;
    h_b[i].velocity = velocity;
    h_b[i].acceleration = acceleration;
}

Body *BarnesHutCuda::getBodies()
{
    return h_b;
}

void BarnesHutCuda::readDeviceBodies()
{
    // Copia de la información de los cuerpos de la GPU al host
    CHECK_CUDA_ERROR(hipMemcpy(h_b, d_b, sizeof(Body) * nBodies, hipMemcpyDeviceToHost));
}

void BarnesHutCuda::setup(int sim)
{
    // Inicializa los cuerpos de manera aleatoria (excepto el sol)
    initRandomBodies();

    // Copia la información al dispositivo
    CHECK_CUDA_ERROR(hipMemcpy(d_b, h_b, sizeof(Body) * nBodies, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_node, h_node, sizeof(Node) * nNodes, hipMemcpyHostToDevice));

    // Apply initial SFC ordering if requested
    if (sfcType != NO_SFC && orderTarget == ORDER_BODIES)
    {
        // For bodies, we can do an initial sort after computing the bounding box
        resetCUDA();
        computeBoundingBoxCUDA();
        applySFCOrdering();
    }
}

UpdateTimes BarnesHutCuda::update()
{
    // Eventos para medir cada fase
    hipEvent_t startAll, afterReset, afterBBox, afterOctree, afterForce, afterSFC;
    hipEventCreate(&startAll);
    hipEventCreate(&afterReset);
    hipEventCreate(&afterBBox);
    hipEventCreate(&afterOctree);
    hipEventCreate(&afterForce);
    hipEventCreate(&afterSFC);

    // Marcar el inicio de todas las operaciones GPU de update()
    hipEventRecord(startAll);

    // 1. resetCUDA()
    resetCUDA();
    hipEventRecord(afterReset);

    // 2. computeBoundingBoxCUDA()
    computeBoundingBoxCUDA();
    hipEventRecord(afterBBox);

    // Apply SFC ordering if requested (after bounding box is computed)
    applySFCOrdering();
    hipEventRecord(afterSFC);

    // 3. constructOctreeCUDA()
    constructOctreeCUDA();
    hipEventRecord(afterOctree);

    // 4. computeForceCUDA()
    computeForceCUDA();
    hipEventRecord(afterForce);

    // Sincronizar para asegurarnos de que los kernels terminaron
    hipEventSynchronize(afterForce);

    // Calcular tiempos entre eventos (ms)
    UpdateTimes times;
    hipEventElapsedTime(&times.resetTimeMs, startAll, afterReset);
    hipEventElapsedTime(&times.bboxTimeMs, afterReset, afterBBox);
    hipEventElapsedTime(&times.sfcTimeMs, afterBBox, afterSFC);
    hipEventElapsedTime(&times.octreeTimeMs, afterSFC, afterOctree);
    hipEventElapsedTime(&times.forceTimeMs, afterOctree, afterForce);

    // Liberar eventos
    hipEventDestroy(startAll);
    hipEventDestroy(afterReset);
    hipEventDestroy(afterBBox);
    hipEventDestroy(afterSFC);
    hipEventDestroy(afterOctree);
    hipEventDestroy(afterForce);

    // Chequeo de errores de CUDA (si corresponde)
    CHECK_LAST_CUDA_ERROR();

    // Retornar la estructura con los tiempos
    return times;
}

void BarnesHutCuda::getBoundingBox(Vector &min, Vector &max)
{
    CHECK_CUDA_ERROR(hipMemcpy(&min, &h_node[0].topLeftFront, sizeof(Vector), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(&max, &h_node[0].botRightBack, sizeof(Vector), hipMemcpyDeviceToHost));
}

void BarnesHutCuda::initRandomBodies()
{
    // Inicializar la semilla para números aleatorios
    srand(time(NULL));

    double maxDistance = MAX_DIST;
    double minDistance = MIN_DIST;

    Vector centerPos = {CENTERX, CENTERY, CENTERZ};

    // Generar cuerpos (por ejemplo, planetas) de forma dinámica
    for (int i = 0; i < nBodies; ++i)
    {
        // Generar dos números aleatorios uniformes entre 0 y 1
        double u = rand() / (double)RAND_MAX; // Para theta
        double v = rand() / (double)RAND_MAX; // Para phi

        // Theta: ángulo en el plano XY (0 a 2π)
        double theta = 2.0 * M_PI * u;
        // Phi: ángulo desde el eje Z; para una distribución uniforme sobre la esfera,
        // se usa phi = acos(2*v - 1)
        double phi = acos(2.0 * v - 1.0);

        // Generar un radio aleatorio entre minDistance y maxDistance
        double radius = (maxDistance - minDistance) * (rand() / (double)RAND_MAX) + minDistance;

        // Convertir de coordenadas esféricas a cartesianas:
        double x = centerPos.x + radius * sin(phi) * cos(theta);
        double y = centerPos.y + radius * sin(phi) * sin(theta);
        double z = centerPos.z + radius * cos(phi);

        Vector position = {x, y, z};

        // Configurar el cuerpo (por ejemplo, un planeta) como dinámico
        h_b[i].isDynamic = true;
        h_b[i].mass = SUN_MASS;
        h_b[i].radius = SUN_DIA;
        h_b[i].position = position;
        h_b[i].velocity = {0.0, 0.0, 0.0};
        h_b[i].acceleration = {0.0, 0.0, 0.0};
    }
}

void BarnesHutCuda::debugPrintDeviceBodies()
{
    // Crear un vector temporal en host para almacenar los cuerpos.
    std::vector<Body> tempBodies(nBodies);

    // Copiar los cuerpos desde el dispositivo (d_b) al host.
    hipError_t err = hipMemcpy(tempBodies.data(), d_b, nBodies * sizeof(Body), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        std::cerr << "Error al copiar cuerpos desde el dispositivo: "
                  << hipGetErrorString(err) << std::endl;
        return;
    }

    std::cout << "===== Cuerpos en el Dispositivo =====" << std::endl;
    for (int i = 0; i < nBodies; i++)
    {
        std::cout << "Cuerpo " << i << ":" << std::endl;
        std::cout << "\tPosición: ("
                  << tempBodies[i].position.x << ", "
                  << tempBodies[i].position.y << ", "
                  << tempBodies[i].position.z << ")" << std::endl;
        std::cout << "\tVelocidad: ("
                  << tempBodies[i].velocity.x << ", "
                  << tempBodies[i].velocity.y << ", "
                  << tempBodies[i].velocity.z << ")" << std::endl;
        std::cout << "\tAceleración: ("
                  << tempBodies[i].acceleration.x << ", "
                  << tempBodies[i].acceleration.y << ", "
                  << tempBodies[i].acceleration.z << ")" << std::endl;
        std::cout << "\tMasa: " << tempBodies[i].mass
                  << ", Radio: " << tempBodies[i].radius
                  << ", Dinámico: " << (tempBodies[i].isDynamic ? "Sí" : "No")
                  << std::endl;
    }
}

void BarnesHutCuda::debugPrintTree()
{
    if (nNodes <= 0)
    {
        std::cout << "El árbol Barnes-Hut está vacío. nNodes = " << nNodes << std::endl;
        return;
    }

    // Crear un vector temporal en host para almacenar los nodos del árbol.
    std::vector<Node> tempNodes(nNodes);

    // Copiar los nodos desde el dispositivo (d_node) al host.
    hipError_t err = hipMemcpy(tempNodes.data(), d_node, nNodes * sizeof(Node), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        std::cerr << "Error al copiar nodos desde el dispositivo: "
                  << hipGetErrorString(err) << std::endl;
        return;
    }

    // Obtener el arreglo de cuerpos del host.
    Body *hostBodies = getBodies(); // Se asume que getBodies() devuelve el arreglo actualizado

    std::cout << "===== Árbol Barnes-Hut =====" << std::endl;
    std::cout << "Cantidad de nodos: " << nNodes << std::endl;

    // Imprimir el bounding box global usando el nodo raíz (índice 0)
    std::cout << "Bounding Box Global (nodo raíz):" << std::endl;
    std::cout << "\tTop Left Front: ("
              << tempNodes[0].topLeftFront.x << ", "
              << tempNodes[0].topLeftFront.y << ", "
              << tempNodes[0].topLeftFront.z << ")" << std::endl;
    std::cout << "\tBot Right Back: ("
              << tempNodes[0].botRightBack.x << ", "
              << tempNodes[0].botRightBack.y << ", "
              << tempNodes[0].botRightBack.z << ")" << std::endl;

    // Recorrer e imprimir cada nodo
    for (int i = 0; i < nNodes; i++)
    {
        std::cout << "Nodo " << i << ":" << std::endl;
        std::cout << "\tTop Left Front: ("
                  << tempNodes[i].topLeftFront.x << ", "
                  << tempNodes[i].topLeftFront.y << ", "
                  << tempNodes[i].topLeftFront.z << ")" << std::endl;
        std::cout << "\tBot Right Back: ("
                  << tempNodes[i].botRightBack.x << ", "
                  << tempNodes[i].botRightBack.y << ", "
                  << tempNodes[i].botRightBack.z << ")" << std::endl;
        std::cout << "\tCentro de Masa: ("
                  << tempNodes[i].centerMass.x << ", "
                  << tempNodes[i].centerMass.y << ", "
                  << tempNodes[i].centerMass.z << ")" << std::endl;
        std::cout << "\tMasa Total: " << tempNodes[i].totalMass << std::endl;
        std::cout << "\tEs Hoja: " << (tempNodes[i].isLeaf ? "Sí" : "No") << std::endl;
        std::cout << "\tRango de cuerpos: inicio = " << tempNodes[i].start
                  << ", fin = " << tempNodes[i].end << std::endl;

        if (sfcType != NO_SFC)
        {
            std::cout << "\tSFC Code: " << tempNodes[i].sfcCode << std::endl;
        }

        // Si el nodo es hoja y tiene un rango válido de cuerpos, imprimir los cuerpos contenidos.
        if (tempNodes[i].isLeaf && tempNodes[i].start != -1 && tempNodes[i].end != -1)
        {
            std::cout << "\tCuerpos en este nodo:" << std::endl;
            for (int j = tempNodes[i].start; j <= tempNodes[i].end; j++)
            {
                // Imprimir detalles básicos del cuerpo; puedes ampliar la información si lo deseas.
                std::cout << "\t\tCuerpo " << j << ": Posición ("
                          << hostBodies[j].position.x << ", "
                          << hostBodies[j].position.y << ", "
                          << hostBodies[j].position.z << ")";
                std::cout << ", Masa " << hostBodies[j].mass;
                std::cout << ", Radio " << hostBodies[j].radius;
                std::cout << ", Dinámico: " << (hostBodies[j].isDynamic ? "Sí" : "No") << std::endl;
            }
        }
    }
}